#include "hip/hip_runtime.h"
#include "td_ins-td_del-runtime.h"
#include "sequential_heap.h"
#include <bits/stdc++.h>
#include <ctime>
#include<assert.h>
#include <unistd.h>
using namespace std;

int *arr;
int *received_arr;

void test() {
    int n = NUMBER_OF_NODES;
    int heap_capacity = n * BATCH_SIZE;
    arr = new int[heap_capacity];
    received_arr = new int[heap_capacity];

    // initialise array for input
    for(int i = 0 ; i < heap_capacity; i++)
        arr[i] = rand() % 5000000;
    
    // initialise input array for device
    int *d_arr;
    gpuErrchk( hipMalloc((void**)&d_arr, heap_capacity * sizeof(int)));
    gpuErrchk( hipMemcpy(d_arr, (arr) , heap_capacity * sizeof(int), hipMemcpyHostToDevice)); 

    int *d_arr_rec;
    gpuErrchk( hipMalloc((void**)&d_arr_rec, heap_capacity * sizeof(int)));
    hipDeviceSynchronize();
    std::clock_t c_start = std::clock();

    for(int i = 0; i < n  ; i++) {
        insert_keys(d_arr + i * BATCH_SIZE, BATCH_SIZE);   
    }
    for(int i = 0 ; i < n; i++) {
        delete_keys(d_arr_rec + i * BATCH_SIZE);
    }
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    std::clock_t c_end = std::clock();
    long double time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "GPU time used: " << time_elapsed_ms << " ms\n";

    priority_queue<int> pq;
    c_start = std::clock();
    // for(int i = 1; i < n ; i++)
    // {
    //     for(int j = i * BATCH_SIZE; j < (i+1) * BATCH_SIZE ; j++)
    //     {
    //         pq.push(arr[j]);
    //     }
    // }
    // while(pq.size()!=0) {
    //     pq.pop();
    // }
    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "CPU-STL time used: " << time_elapsed_ms << " ms\n";

    // CPU_Heap my_heap(heap_capacity);
    c_start = std::clock();
    // for(int i = 1; i < n ; i++)
    // {
    //     for(int j = i * BATCH_SIZE; j < (i+1) * BATCH_SIZE ; j++)
    //     {
    //         my_heap.push(arr[j]);
    //     }
    // }
    // while(not my_heap.is_empty()) {
    //     my_heap.pop();
    // }
    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "CPU my heap time used: " << time_elapsed_ms << " ms\n";

    gpuErrchk( hipMemcpy(received_arr, d_arr_rec, heap_capacity * sizeof(int), hipMemcpyDeviceToHost));

    // verify
    sort(arr, arr + heap_capacity);
    bool correct = 1;
    for(int i = 0 ; i < heap_capacity ; i++) {
        if (arr[i] != received_arr[i]) {
            correct = 0;
            break;
        }
    }

    cout << ((correct)?"Success\n":"Failed!\n");
}

int main()
{
    heap_init();
    // srand(0);
    srand(time(NULL));
    test();
    
}