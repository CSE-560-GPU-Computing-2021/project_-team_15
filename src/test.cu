#include "hip/hip_runtime.h"
#include "td_ins-td_del.h"
#include "sequential_heap.h"
#include <bits/stdc++.h>
#include <ctime>

int arr[HEAP_CAPACITY];
int received_arr[HEAP_CAPACITY];
Heap *b = (Heap*)malloc(sizeof(Heap));
int number_of_streams = 20;
CPU_Heap my_heap(HEAP_CAPACITY);

void test_insertion()
{
    int n = HEAP_CAPACITY / BATCH_SIZE - 1;

    // create random elements for insertion
    for(int i = 0 ; i < HEAP_CAPACITY; i++)
        arr[i] = rand() % 10000000;
    
    std::clock_t c_start = clock(), c_end = clock();
    std::clock_t c_start_mem = clock(), c_end_mem = clock();

    // create multiple streams with non blocking flag
    hipStream_t stream[number_of_streams];
    for(int i = 1 ; i < number_of_streams ; i++)
        hipStreamCreateWithFlags(&(stream[i]), hipStreamNonBlocking);
    
    // create copy of arr on device
    int *d_arr;
    gpuErrchk( hipMalloc((void**)&d_arr, HEAP_CAPACITY * sizeof(int)));
    gpuErrchk( hipMemcpy(d_arr, arr , HEAP_CAPACITY * sizeof(int), hipMemcpyHostToDevice));
    c_end_mem = clock();
    long double time_elapsed_ms_mem = 1000.0 * (c_end_mem - c_start_mem) / CLOCKS_PER_SEC;

    c_start = std::clock();
    for(int i = 1; i <n  ; i++)
        td_insertion<<<1, BLOCK_SIZE,0, stream[i%(number_of_streams - 1) + 1]>>>(d_arr + i*BATCH_SIZE, BATCH_SIZE, d_heap_locks, d_partial_buffer, d_heap, i);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    c_end = std::clock();

    long double time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "GPU time for invocations: " << time_elapsed_ms << " ms\n";

    c_start_mem = clock();
    gpuErrchk( hipMemcpy(b, d_heap, sizeof(Heap), hipMemcpyDeviceToHost));
    c_end_mem = clock();
    time_elapsed_ms_mem += 1000.0 * (c_end_mem - c_start_mem) / CLOCKS_PER_SEC;
    std::cout << "GPU time for memcpy: " << time_elapsed_ms_mem << " ms\n";
    std::cout << "Total GPU time: " << time_elapsed_ms_mem + time_elapsed_ms << " ms\n";

    priority_queue<int, vector<int>, greater<int>> pq;
    c_start = std::clock();
    for(int i = 1; i < n ; i++)
        for(int j = i * BATCH_SIZE; j < (i+1) * BATCH_SIZE ; j++)
            pq.push(arr[j]);
    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "CPU STL-Heap time: " << time_elapsed_ms << " ms\n";


    c_start = std::clock();
    for(int i = 1; i < n ; i++)
        for(int j = i * BATCH_SIZE; j < (i+1) * BATCH_SIZE ; j++)
            my_heap.push(arr[j]);
    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    std::cout << "CPU my-heap time used: " << time_elapsed_ms << " ms\n";

    // verify correctness
    bool correct = 1;
    for(int i = BATCH_SIZE ; i < 2*BATCH_SIZE ; i++)
    {
        int x = my_heap.pop();
        int y = pq.top();
        pq.pop();
        if (x != y){
            correct = 0;
        }
    }
    // correctness verified

    cout << ((correct)?"Success\n":"Failed!\n");
}

int main(int argc, char *argv[])
{   
    heap_init();
    test_insertion();
}