#include "hip/hip_runtime.h"
/*
 * Author: Shubham Mittal
 * IIITD, 2018101
 * GPU Project: Accelerating Concurrent Heap on GPUs
*/
#include "td_ins-td_del.h"

__device__ int get_lock_state(int node_idx, 
                              int *heap_locks)
{
    return heap_locks[node_idx];
}

__device__ void take_lock(int *lock, 
                          int lock_state_1, 
                          int lock_state_2)
{
    while (atomicCAS(lock, lock_state_1, lock_state_2) != lock_state_1);
}

__device__ int try_lock(int *lock, 
                        int lock_state_1, 
                        int lock_state_2)
{
    return atomicCAS(lock, lock_state_1, lock_state_2);
}

__device__ void release_lock(int *lock, 
                             int lock_state_1, 
                             int lock_state_2)
{
    atomicCAS(lock, lock_state_1, lock_state_2);
}

__device__ void bitonic_sort(int *arr, 
                             int size)
{
    // assuming size = power of 2
    int my_thread_id = threadIdx.x;
    int maximum = 0, minimum = 0, other_idx = 0, i = 2, j = 2;
    // parity of batch number will tell which operation to perform in that batch:
    // min/max for even/odd respectively
    int my_batch_number = my_thread_id >> 1;
    for (i = 2; i <= size ; i <<= 1, my_batch_number >>= 1)
    {
        for(j = i; j >= 2 ; j >>= 1)
        {
            int steps_to_look_ahead = j >> 1;
            // only first half of any batch can be active
            if ((my_thread_id % j < steps_to_look_ahead) && (my_thread_id < size)) 
            {
                other_idx = my_thread_id + steps_to_look_ahead;
                minimum = min(arr[my_thread_id], arr[other_idx]);
                maximum = max(arr[my_thread_id], arr[other_idx]); // chances of improvement by using minimum to find max
                if(my_batch_number & 1)
                {
                    arr[my_thread_id] = maximum;
                    arr[other_idx] = minimum;
                }
                else
                {
                    arr[my_thread_id] = minimum;
                    arr[other_idx] = maximum;
                }
            }
            __syncthreads();
        }
    }
}


__device__ void copy_shared_to_global(int *shared, int from_shared_idx1, int to_shared_idx2, int *global, int from_global_idx1)
{
    int my_thread_id = threadIdx.x;
    int n = to_shared_idx2 - from_shared_idx1;
    if(my_thread_id < n)
    {
        global[from_global_idx1 + my_thread_id] = shared[from_shared_idx1 + my_thread_id];
    }
    __syncthreads();
}

__device__ void copy_global_to_shared(int *global, int from_global_idx1, int to_global_idx2, int *shared, int from_shared_idx1)
{
    int my_thread_id = threadIdx.x;
    int n = to_global_idx2 - from_global_idx1;
    if(my_thread_id < n)
    {
        shared[from_shared_idx1 + my_thread_id] = global[from_global_idx1 + my_thread_id];
    }
    __syncthreads();
}

__global__ void heap_init(Heap *heap, 
                          Partial_Buffer *partial_buffer)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < HEAP_CAPACITY)
    {
        heap -> arr[index] = INT_MAX;
    }
    if(index < PARTIAL_BUFFER_CAPACITY)
    {
        partial_buffer -> arr[threadIdx.x] = INT_MAX;   
    }
}

__global__ void td_insertion(int *items_to_be_inserted, 
                             int number_of_items_to_be_inserted, 
                             int *heap_locks, 
                             Partial_Buffer *partial_buffer,
                            Heap *heap)
{
    /*
     * number_of_items_to_be_inserted <= BATCH_SIZE
    */
    __shared__ int items_to_be_inserted_shared_mem[BATCH_SIZE];
    __shared__ int array_to_be_merged_shared_mem[BATCH_SIZE];
    __shared__ int merged_array_shared_mem[BATCH_SIZE << 1];

    int my_thread_id = threadIdx.x;
    copy_global_to_shared(items_to_be_inserted, 0, number_of_items_to_be_inserted, items_to_be_inserted_shared_mem, 0);
    
    bitonic_sort(items_to_be_inserted_shared_mem, number_of_items_to_be_inserted);

    int root_node_idx = 0;
    if (my_thread_id == 0)
        take_lock(&heap_locks[root_node_idx], AVAILABLE, INUSE);

    if (partial_buffer -> size + number_of_items_to_be_inserted >= BATCH_SIZE)
    {

    }
    else
    {
        release_lock(&heap_locks[root_node_idx], AVAILABLE, INUSE);
        // return;
    }
    if (my_thread_id < number_of_items_to_be_inserted)
    {
        items_to_be_inserted[my_thread_id] = items_to_be_inserted_shared_mem[my_thread_id] ;
    }
}

__host__ void heap_init()
{
    gpuErrchk( hipMalloc(&d_partial_buffer, sizeof(Partial_Buffer)));
    gpuErrchk( hipMalloc(&d_heap, sizeof(Heap))); // need to fill with INT_MAX

    heap_init<<<ceil(HEAP_CAPACITY/1024.0), 1024>>>(d_heap, d_partial_buffer);

    gpuErrchk( hipMalloc((void**)&d_heap_locks, NUMBER_OF_NODES * sizeof(int)) );
    gpuErrchk( hipMemset(d_heap_locks, AVAILABLE, NUMBER_OF_NODES * sizeof(int)) );
}