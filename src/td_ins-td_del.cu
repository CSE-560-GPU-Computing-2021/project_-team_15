#include "hip/hip_runtime.h"
/*
 * Author: Shubham Mittal
 * IIITD, 2018101
 * GPU Project: Accelerating Concurrent Heap on GPUs
*/
#include "td_ins-td_del.h"


__device__ int get_lock_state(int node_idx, int *heap_locks) {
    return heap_locks[node_idx];
}


__device__ void take_lock(int *lock, int lock_state_1, int lock_state_2) {
    while (atomicCAS(lock, lock_state_1, lock_state_2) != lock_state_1);
}


__device__ int try_lock(int *lock, int lock_state_1, int lock_state_2) {
    return atomicCAS(lock, lock_state_1, lock_state_2);
}


__device__ void release_lock(int *lock, int lock_state_1, int lock_state_2) {
    atomicCAS(lock, lock_state_1, lock_state_2);
}


__global__ void heap_init(Heap *heap, Partial_Buffer *partial_buffer) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    heap -> global_id = 1;
    heap -> global_idx = 1;
    heap -> size = 0;
    partial_buffer -> size = 0;
    if (index < HEAP_CAPACITY) {
        heap -> arr[index] = INT_MAX;
    }
    if (index < PARTIAL_BUFFER_CAPACITY) {
        partial_buffer -> arr[threadIdx.x] = INT_MAX;   
    }
}


__device__ int bit_reversal(int n, int level) {
    if (n <= 3)
        return n;

    int ans = 1 << (level--);
    while(n != 1) {
        ans += (n & 1) << (level--);
        n >>= 1;
    }
    return ans;

}
__device__ void copy_arr1_to_arr2(int *arr1, int from_arr1_idx1, int to_arr1_idx2, int *arr2, int from_arr2_idx1) {
    int my_thread_id = threadIdx.x;
    int n = to_arr1_idx2 - from_arr1_idx1;
    if (my_thread_id < n) {
        arr2[from_arr2_idx1 + my_thread_id] = arr1[from_arr1_idx1 + my_thread_id];
    }
}


__device__ void memset_arr(int *arr, int from_arr_idx1, int to_arr_idx2, int val) {
    int my_thread_id = threadIdx.x;
    int n = to_arr_idx2 - from_arr_idx1;
    if (my_thread_id < n) {
        arr[from_arr_idx1 + my_thread_id] = val;
    }
    __syncthreads();
}


__device__ void bitonic_sort(int *arr, int size) {
    // assuming size = power of 2
    int my_thread_id = threadIdx.x;
    int maximum = 0, minimum = 0, other_idx = 0, i = 2, j = 2;
    // parity of batch number will tell which operation to perform in that batch:
    // min/max for even/odd respectively
    int my_batch_number = my_thread_id >> 1;
    for (i = 2; i <= size ; i <<= 1, my_batch_number >>= 1) {
        for (j = i; j >= 2 ; j >>= 1) {
            int steps_to_look_ahead = j >> 1;
            // only first half of any batch can be active
            if ((my_thread_id % j < steps_to_look_ahead) && (my_thread_id < size)) 
            {
                other_idx = my_thread_id + steps_to_look_ahead;
                minimum = min(arr[my_thread_id], arr[other_idx]);
                maximum = max(arr[my_thread_id], arr[other_idx]); // chances of improvement by using minimum to find max
                if (my_batch_number & 1) {
                    arr[my_thread_id] = maximum;
                    arr[other_idx] = minimum;
                }
                else {
                    arr[my_thread_id] = minimum;
                    arr[other_idx] = maximum;
                }
            }
            __syncthreads();
        }
    }
}


__device__ int binary_search(int *arr1, int high, int search, bool consider_equality) {
    int low = 0, mid = 0;
    int ans = high;
    while (low <= high)
    {
        mid = (low + high) >> 1;
        if (arr1[mid] >= search and consider_equality) {
            ans = mid;
            high = mid - 1;
        }
        else if (arr1[mid] > search) {
            ans = mid;
            high = mid - 1;
        }
        else
            low = mid + 1;
    }
    return ans;
}


__device__ void merge_and_sort(int *arr1, int idx1, int *arr2, int idx2, int *merged_arr) {
    int my_thread_id = threadIdx.x;

    if (my_thread_id < idx1) {
        merged_arr[my_thread_id + binary_search(arr2, idx2, arr1[my_thread_id], 1)] = arr1[my_thread_id];
    }

    if (my_thread_id < idx2) {
        merged_arr[my_thread_id + binary_search(arr1, idx1, arr2[my_thread_id], 0)] = arr2[my_thread_id];
    }

    __syncthreads();
    
}


__global__ void td_insertion(int *items_to_be_inserted, int number_of_items_to_be_inserted, int *heap_locks, Partial_Buffer *partial_buffer, Heap *heap, int my_id) {
    /*
     * number_of_items_to_be_inserted <= BATCH_SIZE
    */
    const int double_batch_size = BATCH_SIZE << 1;
    __shared__ int items_to_be_inserted_shared_mem[BATCH_SIZE];
    __shared__ int array_to_be_merged_shared_mem[BATCH_SIZE];
    __shared__ int merged_array_shared_mem[double_batch_size];

    int my_thread_id = threadIdx.x;

    // memset_arr(items_to_be_inserted_shared_mem, number_of_items_to_be_inserted, BATCH_SIZE);
    // memset_arr(array_to_be_merged_shared_mem, 0, BATCH_SIZE);
    // memset_arr(merged_array_shared_mem, 0, BATCH_SIZE << 1);

    // copy keys to be inserted in shared memory
    copy_arr1_to_arr2(items_to_be_inserted, 0, number_of_items_to_be_inserted, items_to_be_inserted_shared_mem, 0);
    __syncthreads();

    // sort the keys to be inserted
    bitonic_sort(items_to_be_inserted_shared_mem, number_of_items_to_be_inserted);

    // take root node lock
    if (my_thread_id == MASTER_THREAD){
        while(atomicCAS(&(heap -> global_id), my_id, 0) != my_id);
        take_lock(&heap_locks[ROOT_NODE_IDX], AVAILABLE, INUSE);
        heap -> global_id = my_id + 1;
    }

    __syncthreads();

    // copy partial buffer into shared memory
    copy_arr1_to_arr2(partial_buffer -> arr, 0, partial_buffer -> size, array_to_be_merged_shared_mem, 0);
    __syncthreads();

    // merge partial buffer and keys to be inserted
    merge_and_sort(items_to_be_inserted_shared_mem, number_of_items_to_be_inserted, \
            array_to_be_merged_shared_mem, partial_buffer -> size, merged_array_shared_mem);
    
    int combined_size = partial_buffer -> size + number_of_items_to_be_inserted;
    if (combined_size >= BATCH_SIZE) {

        // copy batch_size into insertion key list
        copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, items_to_be_inserted_shared_mem, 0);

        // copy rest over in partial buffer and update its size
        copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, combined_size - BATCH_SIZE, partial_buffer -> arr, 0);
        __syncthreads();

        // update partial buffer size
        if (my_thread_id == MASTER_THREAD)
            partial_buffer -> size = combined_size - BATCH_SIZE;
    }
    else {
        if (heap -> size == 0) {
            // transfer all keys in partial buffer
            copy_arr1_to_arr2(merged_array_shared_mem, 0, combined_size, partial_buffer -> arr, 0);
            if (my_thread_id == MASTER_THREAD)
                partial_buffer -> size = combined_size;
        }
        else {
            // copy partial buffer into shared array
            copy_arr1_to_arr2(merged_array_shared_mem, 0, combined_size, array_to_be_merged_shared_mem, 0);
            
            // copy root node into shared memory
            copy_arr1_to_arr2(heap -> arr, ROOT_NODE_IDX * BATCH_SIZE, ROOT_NODE_IDX * BATCH_SIZE + BATCH_SIZE, items_to_be_inserted_shared_mem, 0);
            __syncthreads();

            // merge partial buffer with root node
            merge_and_sort(items_to_be_inserted_shared_mem, BATCH_SIZE, array_to_be_merged_shared_mem, combined_size, merged_array_shared_mem);
        
            // copy back to root node
            copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, heap -> arr, ROOT_NODE_IDX * BATCH_SIZE);

            // copy to partial buffer
            copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, BATCH_SIZE + combined_size, partial_buffer -> arr, 0);

            if (my_thread_id == MASTER_THREAD)
                partial_buffer -> size = combined_size;
        }
        if (my_thread_id == MASTER_THREAD)
            release_lock(&heap_locks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        return;
    }

    if (my_thread_id == MASTER_THREAD)
        (heap -> size += 1);
    __syncthreads();

    int tar = heap -> size, level = __log2f(tar); // may have floating point error, default to -1
    // int dummy_tar = tar;
    // while(dummy_tar) {
    //     level++;
    //     dummy_tar >>= 1;
    // }

    tar = bit_reversal(tar, level);
    
    // take lock on target node 
    if (tar != ROOT_NODE_IDX) {
        if (my_thread_id == MASTER_THREAD) {
            take_lock(&heap_locks[tar], AVAILABLE, INUSE);
        }
        __syncthreads();
    }

    int low = 0, high = 0, cur = ROOT_NODE_IDX;;
    while (cur != tar) {
        if (get_lock_state(tar, heap_locks) == MARKED) { // next delete operation can cooperate with current insert operation
            break;
        }
        
        low = cur * BATCH_SIZE;
        high = low + BATCH_SIZE;
        // copy current node to shared mem
        copy_arr1_to_arr2(heap -> arr, low, high, array_to_be_merged_shared_mem, 0);
        __syncthreads();

        // merger current batch with insertion list in shared mem
        merge_and_sort(array_to_be_merged_shared_mem, BATCH_SIZE, items_to_be_inserted_shared_mem, BATCH_SIZE, merged_array_shared_mem);

        // copy back to current batch
        copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, heap -> arr, low);

        __syncthreads();
        // copy to insertion list
        copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, double_batch_size, items_to_be_inserted_shared_mem, 0);

        cur = tar >> (--level);

        if(my_thread_id == MASTER_THREAD) {
            if (cur != tar) {
                take_lock(&heap_locks[cur], AVAILABLE, INUSE);
            }
            release_lock(&heap_locks[cur >> 1], INUSE, AVAILABLE);
        }
        __syncthreads();
    }

    if(my_thread_id == MASTER_THREAD) {
        try_lock(&heap_locks[tar], TARGET, INUSE);
    }
    __syncthreads();
    tar = (get_lock_state(tar, heap_locks) == INUSE) ? tar : 1;
    copy_arr1_to_arr2(items_to_be_inserted_shared_mem, 0, BATCH_SIZE, heap -> arr , tar * BATCH_SIZE);

    if(my_thread_id == MASTER_THREAD) {
        if(tar != cur) {
            release_lock(&heap_locks[tar], get_lock_state(tar, heap_locks), AVAILABLE);
        }
        release_lock(&heap_locks[cur], INUSE, AVAILABLE);
    }
}

__global__ void td_delete(int *items_deleted, int *heap_locks, Partial_Buffer *partial_buffer, Heap *heap, int my_id) {
    
    int my_thread_id = threadIdx.x;
    const int double_batch_size = BATCH_SIZE << 1;
    __shared__ int arr1_shared_mem[BATCH_SIZE];
    __shared__ int arr2_shared_mem[BATCH_SIZE];
    __shared__ int arr3_shared_mem[BATCH_SIZE];
    __shared__ int merged_array_shared_mem[double_batch_size];


    // take root node lock
    if (my_thread_id == MASTER_THREAD)
    {
        while(atomicCAS(&(heap -> global_id), my_id, 0) != my_id);
        take_lock(&heap_locks[ROOT_NODE_IDX], AVAILABLE, INUSE);
        heap -> global_id = my_id + 1;
        // printf("%d \n", my_id);
    }
    __syncthreads();

    // heap is empty
    if (heap -> size == 0) {
        if (partial_buffer -> size != 0) {
            copy_arr1_to_arr2(partial_buffer -> arr, 0, partial_buffer -> size, items_deleted + (heap -> global_idx) * BATCH_SIZE, 0);
            __syncthreads();
            if(my_thread_id == MASTER_THREAD) {
                partial_buffer -> size = 0;
                heap -> global_idx += 1;
            }
            __syncthreads();
        }
        if (my_thread_id == MASTER_THREAD) {
            release_lock(&heap_locks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        }
        __syncthreads();
        return;
    }

    // copy root into shared mem arr1 to be used now and later too
    copy_arr1_to_arr2(heap -> arr, ROOT_NODE_IDX * BATCH_SIZE, ROOT_NODE_IDX * BATCH_SIZE + BATCH_SIZE, arr1_shared_mem, 0);
    // copy root node into list of deleted mem
    copy_arr1_to_arr2(arr1_shared_mem, 0, BATCH_SIZE, items_deleted, heap -> global_idx * BATCH_SIZE);
    __syncthreads();
    if(my_thread_id == MASTER_THREAD) {
        heap -> global_idx += 1;
    }
    __syncthreads();
    int tar = heap -> size;

    if (tar == 1) { // WARNING: not written in pseudocode
        if (partial_buffer -> size == 0) {
            if (my_thread_id == MASTER_THREAD)
                heap -> size = 0;
        }
        else {
            copy_arr1_to_arr2(partial_buffer -> arr, 0 , partial_buffer -> size, heap -> arr, ROOT_NODE_IDX * BATCH_SIZE);
            __syncthreads();
            partial_buffer -> size = 0;
        }
        if(my_thread_id == MASTER_THREAD) {
            release_lock(&heap_locks[ROOT_NODE_IDX], INUSE, AVAILABLE);
        }
        __syncthreads();
        return; 
    }

    int level = __log2f(tar);
    tar = bit_reversal(tar, level);
    int cur = 1;
    
    if (my_thread_id == MASTER_THREAD) {
        try_lock(&heap_locks[tar], TARGET, MARKED);
    } 

    __syncthreads(); // necessary so that master thread do not decrement while other threads are initialising tar
    if (my_thread_id == MASTER_THREAD)
        heap -> size -= 1;
    

    if (get_lock_state(tar, heap_locks) == MARKED) {
        while(get_lock_state(tar, heap_locks) != AVAILABLE);
    }
    else {
        if (my_thread_id == MASTER_THREAD) {
            take_lock(&heap_locks[tar], AVAILABLE, INUSE);
        }
        __syncthreads();
        // root node elements are already copied in arr1
        copy_arr1_to_arr2(heap -> arr, tar * BATCH_SIZE, (tar + 1) * BATCH_SIZE, arr1_shared_mem, 0);
        __syncthreads();
        memset_arr(heap -> arr, tar * BATCH_SIZE, (tar + 1) * BATCH_SIZE, INT_MAX);
        __syncthreads();

        if (my_thread_id == MASTER_THREAD) {
            release_lock(&heap_locks[tar], INUSE, AVAILABLE);
        }        
    }

    // copy partial buffer in arr2_shared mem
    copy_arr1_to_arr2(partial_buffer -> arr, 0, partial_buffer -> size, arr2_shared_mem, 0);
    __syncthreads();

    // merge sort partial buffer in arr2 with root node in arr1
    merge_and_sort(arr1_shared_mem, BATCH_SIZE, arr2_shared_mem, partial_buffer -> size, merged_array_shared_mem);

    // put back to partial buffer since never used
    copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, BATCH_SIZE + partial_buffer -> size, partial_buffer -> arr, 0);

    // copy back to arr1
    copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, arr1_shared_mem, 0);

    int left = 0, right = 0;
    int largest_left = 0, largest_right = 0;
    while(1) {
        
        if((cur << 1) >= NUMBER_OF_NODES) {
            break; // same code after while loop
        }

        left = cur << 1;
        right = left + 1;

        if(my_thread_id == MASTER_THREAD) {
            // take lock on left and right child with mainatining order to avoid possible deadlock
            take_lock(&heap_locks[left], AVAILABLE, INUSE);
            take_lock(&heap_locks[right], AVAILABLE, INUSE);
        }
        __syncthreads();


        copy_arr1_to_arr2(heap -> arr, left * BATCH_SIZE, (left * BATCH_SIZE) + BATCH_SIZE, arr2_shared_mem, 0);
        copy_arr1_to_arr2(heap -> arr, right * BATCH_SIZE, (right * BATCH_SIZE) + BATCH_SIZE, arr3_shared_mem, 0);
        __syncthreads();

        largest_left = arr2_shared_mem[BATCH_SIZE - 1];
        largest_right = arr3_shared_mem[BATCH_SIZE - 1];

        merge_and_sort(arr2_shared_mem, BATCH_SIZE, arr3_shared_mem, BATCH_SIZE, merged_array_shared_mem);

        // swap left right to avoid code duplication
        if(largest_left > largest_right) {
            int temp = left;
            left = right;
            right = temp;
        }
        
        // now right will be largest element

        copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, double_batch_size, heap -> arr, right * BATCH_SIZE);
        __syncthreads();
        if(my_thread_id == MASTER_THREAD) {
            release_lock(&heap_locks[right], INUSE, AVAILABLE);
        }
        copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, arr2_shared_mem, 0);
        __syncthreads();
        
        // temporary debug comment
        if(arr1_shared_mem[BATCH_SIZE - 1] <= arr2_shared_mem[0]) {
            copy_arr1_to_arr2(arr2_shared_mem, 0, BATCH_SIZE, heap -> arr, left * BATCH_SIZE);
            __syncthreads();
            if(my_thread_id == MASTER_THREAD) {
                release_lock(&heap_locks[left], INUSE, AVAILABLE);
            }
            break;
        }
        merge_and_sort(arr1_shared_mem, BATCH_SIZE, arr2_shared_mem, BATCH_SIZE, merged_array_shared_mem);
        
        copy_arr1_to_arr2(merged_array_shared_mem, 0, BATCH_SIZE, heap -> arr, cur * BATCH_SIZE);
        __syncthreads();

       
        if(my_thread_id == MASTER_THREAD) {
            release_lock(&heap_locks[cur], INUSE, AVAILABLE);
        }
        copy_arr1_to_arr2(merged_array_shared_mem, BATCH_SIZE, double_batch_size, arr1_shared_mem, 0);
        cur = left;
        __syncthreads();
    }

    // copy current array to global heap before releasing lock
    copy_arr1_to_arr2(arr1_shared_mem, 0, BATCH_SIZE, heap -> arr, cur * BATCH_SIZE);
    __syncthreads();

    if(my_thread_id == MASTER_THREAD) {
        release_lock(&heap_locks[cur], INUSE, AVAILABLE);
    }

}
__host__ void heap_init() {
    gpuErrchk( hipMalloc(&d_partial_buffer, sizeof(Partial_Buffer)));
    gpuErrchk( hipMalloc(&d_heap, sizeof(Heap))); // need to fill with INT_MAX

    heap_init<<<ceil(HEAP_CAPACITY / 1024.0), 1024>>>(d_heap, d_partial_buffer);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    gpuErrchk( hipMalloc((void**)&d_heap_locks, NUMBER_OF_NODES * sizeof(int)) );
    gpuErrchk( hipMemset(d_heap_locks, AVAILABLE, NUMBER_OF_NODES * sizeof(int)) );
}
